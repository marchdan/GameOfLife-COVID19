#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>

#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<math.h>

//Declare all needed extern variables and functions

// Result from last compute of world.
extern unsigned char *g_resultData;

// Current state of world. 
extern unsigned char *g_data;

// Current width of world.
extern size_t g_worldWidth;

/// Current height of world.
extern size_t g_worldHeight;

/// Current data length (product of width and height)
extern size_t g_dataLength;  // g_worldWidth * g_worldHeight

// Timing variables
extern size_t s_time;
extern size_t e_time;

//Global storage of total casses and deaths
extern unsigned int totalCases;
extern unsigned int totalDeaths;


extern "C" //Extern functions in C for understandability by mpi file
{
    void initMaster( int myrank, int numranks, unsigned int pattern, size_t worldSize, size_t caseSeed, size_t deathSeed );
    bool kernalLaunch(int myrank, int numranks, 
                    unsigned char** d_data, unsigned char** d_resultData,
                    size_t worldWidth, size_t worldHeight, size_t iterationsCount, ushort threadsCount, size_t pattern,
                    unsigned int infectRate, unsigned int deathRate, unsigned char** recv);
    
    void recvData(size_t myrank, size_t numranks, size_t worldLength, unsigned char** *recv);
    void sendData(size_t myrank, size_t numranks, size_t worldLength, unsigned char* sData);
    void exportStats(unsigned char** data, int myrank, int numranks, int day);


    void freeData();    
    void finishCuda();
}


// Initialize each space in the data grids after initialization
static inline void gol_initData( unsigned char fill )
{
    for(int i = 0; i < g_dataLength; i++){
        g_data[i] = fill; //Fill the grid with the given fill number (0 or 1)
        g_resultData[i] = 0; //Fill result grid with default zero values
    }
}

static inline void gol_initEveryOther( size_t worldWidth, size_t worldHeight, size_t infected, size_t numranks, unsigned int* rands )
{
    size_t local = infected;
    size_t num_rows = local/(worldWidth/2); //Number of rows with infected people
    size_t start = (worldHeight/2) - (num_rows/2); //Row to start on
    size_t current = start*worldWidth; //Starting cell

    //Global variable init
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;
    
    //Allocate data for mian grid and result grid
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));   

    gol_initData(0); //Initalize grid to zero for input of argv[1] = 0

    while(local > 0 && current < g_dataLength){ //Initialize every other spot in the middle of the grid
        g_data[current] = (rands[current]%20)+1;
        local-=1; current+=2;
    }
}

static inline void gol_initClustered( size_t worldWidth, size_t worldHeight, size_t infected, size_t numranks, unsigned int* rands )
{   
    size_t local = infected;
    size_t current = 0;
    size_t spacing = 0;
    size_t clusters = local/4;
    unsigned int i;

    // Set all global values for later use
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    //Allocate data for main grid and result grid
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));

    gol_initData(0); //Initialze main grid to one for input of argv[1] = 1
    
    current = g_worldWidth+1;

    spacing = (g_dataLength-(2*g_worldWidth))/clusters;

    while(local>3){ //Initiate maximum spacing between groups of 4
        if(current&g_worldWidth == 0){ current++;}

        if(current%g_worldWidth < g_worldWidth-5){
            for(i = 0; i < 4; i++, local--, current++){
                g_data[current] = (rands[current]%20)+1;
            }
            current+= (spacing/2);
        }
        else{
            while(current%g_worldWidth > 0){
                current++;
            }
        }
    }

    while(local > 0){
        if(current >= g_dataLength){
            current = 1;
        }

        if(g_data[current-1] == 0 && g_data[current] == 0 && g_data[current+1] == 0){
            g_data[current] = (rands[current]%20)+1;
        }

        local--; current += 3;
    }
}

static inline void gol_initMiddle( size_t worldWidth, size_t worldHeight, size_t infected, size_t numranks, unsigned int* rands )
{
    size_t local = infected;
    size_t dim = sqrt(local);
    size_t start = (worldHeight/2) - (dim/2); //Row to start on
    size_t current = (start*worldWidth)+((worldWidth/2)-(dim/2)); //Starting cell

    int i, j, begin;

    // Set all global values for later use
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // Allocate data for main grid and result grid
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));

    gol_initData(0); //Initialize grids to zero

    // Create clustered square in the middle of the grid
    for( i = 0; i < dim; i++){
        begin = current + (i*worldWidth);
        for( j = begin; j < begin + dim; j++){
            g_data[j] = (rands[j]%20)+1;
        }
    } 
}

void gol_initDistancing( size_t worldWidth, size_t worldHeight, size_t infected, size_t numranks, unsigned int* rands )
{
    size_t local = infected;
    size_t current = worldWidth;

    // Set all global values for later use
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    //Allocate data for main grid and result grid
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));

    gol_initData(0); //Initialize grids to zero

    size_t available_space = g_dataLength-(2* g_worldWidth);
    size_t distance = available_space/local;
    size_t front = distance/2;
    size_t back = distance-front;

    //Initialize with maximum distancing possible
    while(local > 0 && current < g_dataLength){
        current += front;
        g_data[current] = (rands[current]%20)+1; 
        local-=1; current+=back;
    }
}

void gol_initRandom( size_t worldWidth, size_t worldHeight, size_t infected, size_t numranks, unsigned int* rands)
{
    size_t local = infected;
    size_t current = worldWidth;
    
    // Set all global valeus for later use
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    //Allocate data for main grid and result grid
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));

    gol_initData(0); //Initialize grids to zero

    while(local > 0){ //Randomly initialize spaces in the grid
        if(current == g_dataLength){ current = 0; }
        
        if(g_data[current] == 1){
            current++;
            continue;
        }
        else if(rands[current]%100 <= 5){ 
            g_data[current] = (rands[current]%20)+1; 
            local-=1;
        }
        current++;
    }
}

//Initiate the cuda world 
void initMaster( int myrank, int numranks, unsigned int pattern, size_t worldSize, size_t caseSeed, size_t deathSeed )
{
    int cudaDeviceCount = -1;
    hipError_t cE = hipSuccess;

    unsigned int *rands;
    hiprandGenerator_t gen;

    //Initialize cuda devices
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
        cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
        myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }

    totalCases = caseSeed;
    totalDeaths = deathSeed;

    //Alocate and create the host random number generator with curand
    hipMallocManaged( &rands, (worldSize*worldSize * sizeof(unsigned int)));
    for(int i = 0; i < worldSize*worldSize; i++){
        rands[i] = 0;
    }

    hiprandStatus_t stat1, stat2, stat4;
    stat1 = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
    
    stat2 = hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL*(unsigned long long) myrank);
    
    stat4 = hiprandGenerate( gen, rands, worldSize*worldSize);
    hipDeviceSynchronize();
    if(stat1 != HIPRAND_STATUS_SUCCESS){
        printf("ERROR: Generator creation failed\n");
        exit(-1);
    } 
    else if(stat2 != HIPRAND_STATUS_SUCCESS){
        printf("ERROR: Generator seeding failed\n");
        exit(-1);
    }
    else if(stat4 != HIPRAND_STATUS_SUCCESS){
        printf("ERROR: RNG Production failed(%d)\n", stat4);
        exit(-1);
    }
    else if(rands[0] == 0){
        printf("ERROR: Rands did not generate!\n");
        exit(-1);
    }

    printf("Rank %d, of size %ldx%ld, is running GOL kernel on device %d with %ld infected\n", myrank, worldSize, worldSize, (myrank%cudaDeviceCount), caseSeed);
   
    //Initialize the world based on the given pattern
    switch(pattern)
    {
    case 0:
	gol_initEveryOther( worldSize, worldSize, caseSeed, numranks, rands);
	break;
	
    case 1:
	gol_initClustered( worldSize, worldSize, caseSeed, numranks, rands);
	break;
	
    case 2:
	gol_initMiddle( worldSize, worldSize, caseSeed, numranks, rands);
	break;
	
    case 3:
	gol_initDistancing( worldSize, worldSize, caseSeed, numranks, rands);
	break;

    case 4:
	gol_initRandom( worldSize, worldSize, caseSeed, numranks, rands);
	break;

    default:
	printf("Pattern %u has not been implemented \n", pattern);
	exit(-1);
    }

    hipFree(rands);
    hiprandDestroyGenerator(gen);
}

//Swap the information in the given arrays
static inline void gol_swap( unsigned char **pA, unsigned char **pB)
{
    unsigned char *temp = *pA; //Save pA for use later
    *pA = *pB; //Swap pB into pA
    *pB = temp; //Swap pA into pB to complete the swap
}

//Cout the number of alive cells in the current world
__device__ 
static inline unsigned int gol_countAliveCells(unsigned char* data, 
					   size_t x0, size_t x1, size_t x2, 
					   size_t y0, size_t y1,size_t y2, size_t width) 
{
    // Compute the number of infected cells around the current cell
    // Infected cells have a value of 1 - 16, healthy cells have a value of 0
    // Adding up the appropiate cells produces a result containing the number of infected cells

    int BL, L, BR, R, B, UL, UR, U; // Initialize a variable for each direction

    L = data[x0+y1]; // Left
    R = data[x2+y1]; // Right

    BL = data[x0+y2]; //Bottom left
    B = data[x1+y2]; //Bottom
    BR = data[x2+y2]; //Bottom Right

    UL = data[x0+y0]; //Upper left
    U = data[x1+y0]; //Upper
    UR = data[x2+y0]; //Upper right

    int alive = 0; // Computer total number infected
    alive += ((BL < 17 && BL > 0) ? 1 : 0);
    alive += ((B < 17 && B > 0) ? 1 : 0);
    alive += ((BR < 17 && BR > 0) ? 1 : 0);
    alive += ((L < 17 && L > 0) ? 1 : 0);
    alive += ((R < 17 && R > 0) ? 1 : 0);
    alive += ((UL < 17 && UL > 0) ? 1 : 0);
    alive += ((U < 17 && U > 0) ? 1 : 0);
    alive += ((UR < 17 && UR > 0) ? 1 : 0);
    return alive;
}

__device__
unsigned int getDefaultStatus(unsigned int current, int alive, unsigned int *tCases, unsigned int *tDeaths){
    if(current > 0){ // Decide next cell state of a currently living cell.
        if(alive > 4){ // cell doesn't change state unless its healed
            return (current == 1 ? 0 : current);
        }
        else{ //Cell gets one day closer to being healthy
            return current-1;
        }
    }
    else{ //Currently healthy cells
        if(alive < 2){ // Co infection
            return 0;
        }
        else if(alive < 4){ // Minimal infection
            (*tCases)+=1;
            return 14;
        }
        else{ //Full infection
            (*tCases)+=1;
            return 21;
        }
    }
}

__device__
unsigned int getStatsStatus(unsigned int current, int alive, int rate, unsigned int *tCases, unsigned int *tDeaths, hiprandState_t* state){
    if(current > 0){
        return current-1;
    }
    else{
        if(alive == 0){ return 0; }
        while(alive > 0){
            int rand = fabsf(hiprand(state));
            if(rand%100 < rate){ //Spread rate
                (*tCases)++;
                return 21;
            }
            else{
                alive--;
            }
        }
        return 0;
    }
}

__device__
unsigned int getWorstStatus(unsigned int current, int alive, unsigned int *tCases, unsigned int *tDeaths){
    if(current > 0){
        if(alive > 5){
            return (current == 1 ? 0 : current);
        }
        else{
            return current-1;
        }
    }
    else{
        if(alive > 1){
            (*tCases)++;
            return 21;
        }
        else{
            return 0;
        }
    }
}

__device__
unsigned int getBestStatus(unsigned int current, int alive, unsigned int *tCases, unsigned int *tDeaths){
    if(current > 0){
        return current-1;
    }
    else{
        if(alive>6){
            (*tCases)++;
            return 14;
        }
        else{
            return 0;
        }
    }
}

/*
    CUDA kernal for running GOL calculations in parallel with the specificed number of threads/blocks
*/
__global__ void gol_kernal(unsigned int myrank, unsigned int numranks, 
                        unsigned char* d_data,
                        unsigned int worldWidth, unsigned int worldHeight,
                        unsigned char* d_resultData, unsigned int pattern,
                        unsigned int infectRate, unsigned int deathRate, unsigned int *tCases, unsigned int *tDeaths, size_t worldLength)
    {
        unsigned int index, x0, x2, y0, y1, y2, y, x; //Initialize all needed variables for the function

        index = blockIdx.x * blockDim.x + threadIdx.x; //The provided index calculation
        x = index%worldWidth; //The remainder of the index divided by the grid width produced the x component of the index.
        y = index/worldWidth; //The integer answer of index divided by world width provides the y component of the index.
        
        hiprandState_t state;
        
        while(index < worldLength){ //continue as long as the current index is valid in the scope of the provided grid.
            // Provided variable calculations
            hiprand_init(2020, index, 0, &state); //Device rng init
            y0 = ((y+worldHeight-1)%worldHeight)*worldWidth;
            y1 = y*worldWidth;
            y2 = ((y + 1) % worldHeight) * worldWidth;
            x0 = (x + worldWidth - 1) % worldWidth;
            x2 = (x + 1) % worldWidth;

            int alive = gol_countAliveCells(d_data, x0, x, x2, y0, y1, y2, worldWidth); // Retrive the number of current infected cells
            
            if(d_data[index] > 0){ //Chance of cell death
                unsigned int rand = fabsf(hiprand(&state));
                if(rand%100 < deathRate){
                    d_resultData[index] = 0;
                    (*tDeaths)++;
                    index += (blockDim.x * gridDim.x);
                    continue;
                }
            }

            switch(pattern){
                case(0):
                d_resultData[index] = getDefaultStatus(d_data[index], alive, tCases, tDeaths);
                break;

                case(1):
                d_resultData[index] = getStatsStatus(d_data[index], alive, infectRate, tCases, tDeaths, &state);
                break;

                case(2): 
                d_resultData[index] = getWorstStatus(d_data[index], alive, tCases, tDeaths);
                break;

                case(3): 
                d_resultData[index] = getBestStatus(d_data[index], alive, tCases, tDeaths);
                break;

                default:
                printf("Pattern %u has not been implemented \n", pattern);
                return;

            }
            index += (blockDim.x * gridDim.x); //Increase index by one block.
        }
    }

bool kernalLaunch(int myrank, int numranks, 
                    unsigned char** d_data, unsigned char** d_resultData,
                    size_t worldWidth, size_t worldHeight, size_t iterationsCount, ushort threadsCount, size_t pattern,
                    unsigned int infectRate, unsigned int deathRate, unsigned char** recv)
    {
        int i;
        size_t N; //Initialize needed variables
        
        N = worldWidth * worldHeight; //N equals total grid size 
        
        for(i = 0; i < iterationsCount; i++){
            gol_kernal<<<N/threadsCount,threadsCount>>>(myrank, numranks, *d_data, worldWidth, worldHeight, *d_resultData, pattern, infectRate, deathRate, &totalCases, &totalDeaths, N); //Call the Parallel kernel and specify the number of blocks and threads per block.
            gol_swap(d_data, d_resultData);// Swap the current data with the result data
            
            //printf("\tRank %d, Day %d- Cases: %u | Deaths: %u\n", myrank, i, totalCases, totalDeaths);
            
            unsigned char* cpy = *d_data;
            if(numranks > 1){
                hipDeviceSynchronize(); //Synchronize before full exchange
                recvData(myrank, numranks, N, &recv);
                sendData(myrank, numranks, N, cpy);
                hipDeviceSynchronize();//Synchronize before export
            }
            recv[myrank] = cpy;
            exportStats(recv, myrank, numranks, i);
        }
        return true;
    }

void finishCuda(){
    hipDeviceSynchronize(); // Function to synchronize cuda since this call must happen from the MPI code base
}

void freeData(){ //Function to free all CUDA memomory allocated
    hipFree(g_data);
    hipFree(g_resultData);
}
